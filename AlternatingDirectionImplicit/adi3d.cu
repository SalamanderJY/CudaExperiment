#include "hip/hip_runtime.h"
//
// Program to perform ADI time-marching on a regular 3D grid
// Alternating Direction Implicit Method

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include "cutil_inline.h"
#include <stdio.h>

////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////
// define kernel block size for 
////////////////////////////////////////////////////////////////////////

#define BLOCK_X 32
#define BLOCK_Y 8
// device code


__global__ void GPU_adi_rhs(int NX, int NY, int NZ, float lam,
	const float* __restrict__ d_u,
	float* __restrict__ d_du,
	float* __restrict__ d_ax,
	float* __restrict__ d_bx,
	float* __restrict__ d_cx,
	float* __restrict__ d_ay,
	float* __restrict__ d_by,
	float* __restrict__ d_cy,
	float* __restrict__ d_az,
	float* __restrict__ d_bz,
	float* __restrict__ d_cz)
{
	int   i, j, k, indg, active;
	float du, a, b, c;

	int NXM1 = NX - 1;
	int NYM1 = NY - 1;
	int NZM1 = NZ - 1;

#define IOFF 1
#define JOFF NX
#define KOFF NX*NY

	//
	// set up indices for main block
	//

	i = threadIdx.x + blockIdx.x*BLOCK_X;
	j = threadIdx.y + blockIdx.y*BLOCK_Y;
	indg = i + j*NX;

	active = (i<NX) && (j<NY);

	//
	// loop over k-planes
	//

	for (k = 0; k<NZ; k++) {

		//
		// calculate r.h.s. and set a,b,c, coefficients
		//

		if (active) {
			if (i == 0 || i == NXM1 || j == 0 || j == NYM1 || k == 0 || k == NZM1) {
				du = 0.0f;          // Dirichlet b.c.'s
				a = 0.0f;
				b = 1.0f;
				c = 0.0f;
			}
			else {
				du = lam * (d_u[indg - IOFF] + d_u[indg + IOFF]
					+ d_u[indg - JOFF] + d_u[indg + JOFF]
					+ d_u[indg - KOFF] + d_u[indg + KOFF] - 6.0f*d_u[indg]);
				a = -0.5f*lam;
				b = 1.0f + lam;
				c = -0.5f*lam;
			}

			d_du[indg] = du;
			d_ax[indg] = a;
			d_bx[indg] = b;
			d_cx[indg] = c;
			d_ay[indg] = a;
			d_by[indg] = b;
			d_cy[indg] = c;
			d_az[indg] = a;
			d_bz[indg] = b;
			d_cz[indg] = c;

			indg += KOFF;
		}
	}
}

__global__ void GPU_adi_x_float4(int NX, int NY, int NZ,
	const float4* __restrict__ d_a,
	const float4* __restrict__ d_b,
	const float4* __restrict__ d_c,
	float4* __restrict__ d_d) {
	int    i, j, k, indg;
	float  aa, bb, cc, dd, c2[256], d2[256];
	float4 a4, b4, c4, d4;

	//
	// set up indices for main block
	//

	j = threadIdx.x + blockIdx.x*blockDim.x;
	k = threadIdx.y + blockIdx.y*blockDim.y;
	indg = NX*(j + k*NY) / 4;

	if ((j<NY) && (k<NZ)) {

		//
		// forward pass
		//

		a4 = d_a[indg];
		b4 = d_b[indg];
		c4 = d_c[indg];
		d4 = d_d[indg];

		bb = 1.0f / b4.x;
		cc = bb   * c4.x;
		dd = bb   * d4.x;
		c2[0] = cc;
		d2[0] = dd;

		aa = a4.y;
		bb = b4.y - aa*cc;
		dd = d4.y - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4.y;
		dd = bb*dd;
		c2[1] = cc;
		d2[1] = dd;

		aa = a4.z;
		bb = b4.z - aa*cc;
		dd = d4.z - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4.z;
		dd = bb*dd;
		c2[2] = cc;
		d2[2] = dd;

		aa = a4.w;
		bb = b4.w - aa*cc;
		dd = d4.w - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4.w;
		dd = bb*dd;
		c2[3] = cc;
		d2[3] = dd;

		for (i = 4; i<NX; i += 4) {
			indg = indg + 1;
			a4 = d_a[indg];
			b4 = d_b[indg];
			c4 = d_c[indg];
			d4 = d_d[indg];

			aa = a4.x;
			bb = b4.x - aa*cc;
			dd = d4.x - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.x;
			dd = bb*dd;
			c2[i] = cc;
			d2[i] = dd;

			aa = a4.y;
			bb = b4.y - aa*cc;
			dd = d4.y - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.y;
			dd = bb*dd;
			c2[i + 1] = cc;
			d2[i + 1] = dd;

			aa = a4.z;
			bb = b4.z - aa*cc;
			dd = d4.z - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.z;
			dd = bb*dd;
			c2[i + 2] = cc;
			d2[i + 2] = dd;

			aa = a4.w;
			bb = b4.w - aa*cc;
			dd = d4.w - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.w;
			dd = bb*dd;
			c2[i + 3] = cc;
			d2[i + 3] = dd;
		}

		//
		// reverse pass
		//

		d4.w = dd;
		dd = d2[NX - 2] - c2[NX - 2] * dd;
		d4.z = dd;
		dd = d2[NX - 3] - c2[NX - 3] * dd;
		d4.y = dd;
		dd = d2[NX - 4] - c2[NX - 4] * dd;
		d4.x = dd;

		d_d[indg] = d4;

		for (i = NX - 5; i >= 0; i -= 4) {
			indg = indg - 1;
			dd = d2[i] - c2[i] * dd;
			d4.w = dd;
			dd = d2[i - 1] - c2[i - 1] * dd;
			d4.z = dd;
			dd = d2[i - 2] - c2[i - 2] * dd;
			d4.y = dd;
			dd = d2[i - 3] - c2[i - 3] * dd;
			d4.x = dd;

			d_d[indg] = d4;
		}
	}
}


__global__ void GPU_adi_x_float4_2(int NX, int NY, int NZ,
	const float4* __restrict__ d_a,
	const float4* __restrict__ d_b,
	const float4* __restrict__ d_c,
	float4* __restrict__ d_d) {
	int    i, j, k, indg;
	float  aa, bb, cc, dd, c2[256], d2[256];
	float4 a4, b4, c4, d4, a4_2, b4_2, c4_2, d4_2;

	//
	// set up indices for main block
	//

	j = threadIdx.x + blockIdx.x*blockDim.x;
	k = threadIdx.y + blockIdx.y*blockDim.y;
	indg = NX*(j + k*NY) / 4;

	if ((j<NY) && (k<NZ)) {

		//
		// forward pass
		//

		a4 = d_a[indg];
		a4_2 = d_a[indg + 1];

		b4 = d_b[indg];
		b4_2 = d_b[indg + 1];

		c4 = d_c[indg];
		c4_2 = d_c[indg + 1];

		d4 = d_d[indg];
		d4_2 = d_d[indg + 1];

		bb = 1.0f / b4.x;
		cc = bb   * c4.x;
		dd = bb   * d4.x;
		c2[0] = cc;
		d2[0] = dd;

		aa = a4.y;
		bb = b4.y - aa*cc;
		dd = d4.y - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4.y;
		dd = bb*dd;
		c2[1] = cc;
		d2[1] = dd;

		aa = a4.z;
		bb = b4.z - aa*cc;
		dd = d4.z - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4.z;
		dd = bb*dd;
		c2[2] = cc;
		d2[2] = dd;

		aa = a4.w;
		bb = b4.w - aa*cc;
		dd = d4.w - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4.w;
		dd = bb*dd;
		c2[3] = cc;
		d2[3] = dd;

		aa = a4_2.x;
		bb = b4_2.x - aa*cc;
		dd = d4_2.x - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4_2.x;
		dd = bb*dd;
		c2[4] = cc;
		d2[4] = dd;

		aa = a4_2.y;
		bb = b4_2.y - aa*cc;
		dd = d4_2.y - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4_2.y;
		dd = bb*dd;
		c2[5] = cc;
		d2[5] = dd;

		aa = a4_2.z;
		bb = b4_2.z - aa*cc;
		dd = d4_2.z - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4_2.z;
		dd = bb*dd;
		c2[6] = cc;
		d2[6] = dd;

		aa = a4_2.w;
		bb = b4_2.w - aa*cc;
		dd = d4_2.w - aa*dd;
		bb = 1.0f / bb;
		cc = bb*c4_2.w;
		dd = bb*dd;
		c2[7] = cc;
		d2[7] = dd;

		for (i = 8; i<NX; i += 8) {
			indg = indg + 2;

			a4 = d_a[indg];
			a4_2 = d_a[indg + 1];

			__threadfence_block();

			b4 = d_b[indg];
			b4_2 = d_b[indg + 1];

			__threadfence_block();

			c4 = d_c[indg];
			c4_2 = d_c[indg + 1];

			__threadfence_block();

			d4 = d_d[indg];
			d4_2 = d_d[indg + 1];

			aa = a4.x;
			bb = b4.x - aa*cc;
			dd = d4.x - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.x;
			dd = bb*dd;
			c2[i] = cc;
			d2[i] = dd;

			aa = a4.y;
			bb = b4.y - aa*cc;
			dd = d4.y - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.y;
			dd = bb*dd;
			c2[i + 1] = cc;
			d2[i + 1] = dd;

			aa = a4.z;
			bb = b4.z - aa*cc;
			dd = d4.z - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.z;
			dd = bb*dd;
			c2[i + 2] = cc;
			d2[i + 2] = dd;

			aa = a4.w;
			bb = b4.w - aa*cc;
			dd = d4.w - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4.w;
			dd = bb*dd;
			c2[i + 3] = cc;
			d2[i + 3] = dd;

			aa = a4_2.x;
			bb = b4_2.x - aa*cc;
			dd = d4_2.x - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4_2.x;
			dd = bb*dd;
			c2[i + 4] = cc;
			d2[i + 4] = dd;

			aa = a4_2.y;
			bb = b4_2.y - aa*cc;
			dd = d4_2.y - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4_2.y;
			dd = bb*dd;
			c2[i + 5] = cc;
			d2[i + 5] = dd;

			aa = a4_2.z;
			bb = b4_2.z - aa*cc;
			dd = d4_2.z - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4_2.z;
			dd = bb*dd;
			c2[i + 6] = cc;
			d2[i + 6] = dd;

			aa = a4_2.w;
			bb = b4_2.w - aa*cc;
			dd = d4_2.w - aa*dd;
			bb = 1.0f / bb;
			cc = bb*c4_2.w;
			dd = bb*dd;
			c2[i + 7] = cc;
			d2[i + 7] = dd;
		}

		//
		// reverse pass
		//

		d4_2.w = dd;
		dd = d2[NX - 2] - c2[NX - 2] * dd;
		d4_2.z = dd;
		dd = d2[NX - 3] - c2[NX - 3] * dd;
		d4_2.y = dd;
		dd = d2[NX - 4] - c2[NX - 4] * dd;
		d4_2.x = dd;
		dd = d2[NX - 5] - c2[NX - 5] * dd;
		d4.w = dd;
		dd = d2[NX - 6] - c2[NX - 6] * dd;
		d4.z = dd;
		dd = d2[NX - 7] - c2[NX - 7] * dd;
		d4.y = dd;
		dd = d2[NX - 8] - c2[NX - 8] * dd;
		d4.x = dd;

		d_d[indg + 1] = d4_2;
		d_d[indg] = d4;

		for (i = NX - 9; i >= 0; i -= 8) {
			indg = indg - 2;

			dd = d2[i] - c2[i] * dd;
			d4_2.w = dd;
			dd = d2[i - 1] - c2[i - 1] * dd;
			d4_2.z = dd;
			dd = d2[i - 2] - c2[i - 2] * dd;
			d4_2.y = dd;
			dd = d2[i - 3] - c2[i - 3] * dd;
			d4_2.x = dd;
			dd = d2[i - 4] - c2[i - 4] * dd;
			d4.w = dd;
			dd = d2[i - 5] - c2[i - 5] * dd;
			d4.z = dd;
			dd = d2[i - 6] - c2[i - 6] * dd;
			d4.y = dd;
			dd = d2[i - 7] - c2[i - 7] * dd;
			d4.x = dd;

			d_d[indg + 1] = d4_2;
			d_d[indg] = d4;
		}
	}
}

//
// new tri-diagonal solve in x-direction
//

__global__ void GPU_adi_x_new(int NX, int NY, int NZ,
	const float* __restrict__ d_a,
	const float* __restrict__ d_b,
	const float* __restrict__ d_c,
	float* __restrict__ d_d)
{
	int   j, k, indg, t, tm, tp, nt, shift = 0;
	float bbi;
	__shared__  float a[256], c[256], d[256];

	//
	// set up indices for main block
	//

	t = threadIdx.x;
	j = blockIdx.x;
	k = blockIdx.y;
	indg = t + NX*(j + k*NY);

	bbi = 1.0f / d_b[indg];
	a[t] = -bbi * d_a[indg];
	c[t] = -bbi * d_c[indg];
	d[t] = bbi * d_d[indg];

	// forward pass

	tm = 2 * t;
	t = tm + 1;
	tp = tm + 2;

	for (nt = blockDim.x / 2; nt>0; nt >>= 1) {
		shift++;
		__syncthreads();

		if (threadIdx.x < nt) {
			bbi = 1.0f;
			if (tm >= 0) {
				bbi -= a[t] * c[tm];
				d[t] += a[t] * d[tm];
				a[t] = a[t] * a[tm];
			}
			if (tp<NX) {
				bbi -= c[t] * a[tp];
				d[t] += c[t] * d[tp];
				c[t] = c[t] * c[tp];
			}
			bbi = 1.0f / bbi;
			d[t] *= bbi;
			a[t] *= bbi;
			c[t] *= bbi;

			tm = 2 * tm + 1;
			t = 2 * t + 1;
			tp = 2 * tp + 1;
		}
	}

	// reverse pass

	for (; shift>0; shift--) {
		nt = blockDim.x >> shift;
		__syncthreads();

		if (threadIdx.x < nt) {
			tm >>= 1;
			t >>= 1;
			tp >>= 1;
			if (tm >= 0) d[tm] += c[tm] * d[t];
			if (tp<NX) d[tp] += a[tp] * d[t];
		}
	}

	__syncthreads();
	d_d[indg] = d[threadIdx.x];
}


//
// old tri-diagonal solve in x-direction
//

__global__ void GPU_adi_x(int NX, int NY, int NZ,
	const float* __restrict__ d_a,
	const float* __restrict__ d_b,
	const float* __restrict__ d_c,
	float* __restrict__ d_d)
{
	int   i, j, k, indg;
	float aa, bb, cc, dd, c[256], d[256];

	//
	// set up indices for main block
	//

	j = threadIdx.x + blockIdx.x*blockDim.x;  // global indices
	k = threadIdx.y + blockIdx.y*blockDim.y;
	indg = NX*(j + k*NY);

	if ((j<NY) && (k<NZ)) {

		//
		// forward pass
		//

		bb = 1.0f / d_b[indg];
		cc = bb*d_c[indg];
		dd = bb*d_d[indg];
		c[0] = cc;
		d[0] = dd;

		for (i = 1; i<NX; i++) {
			indg = indg + 1;
			aa = d_a[indg];
			bb = d_b[indg] - aa*cc;
			dd = d_d[indg] - aa*dd;
			bb = 1.0f / bb;
			cc = bb*d_c[indg];
			dd = bb*dd;
			c[i] = cc;
			d[i] = dd;
		}

		//
		// reverse pass
		//

		d_d[indg] = dd;

		for (i = NX - 2; i >= 0; i--) {
			indg = indg - 1;
			dd = d[i] - c[i] * dd;
			d_d[indg] = dd;
		}
	}
}


//
// tri-diagonal solve in y-direction
//

__global__ void GPU_adi_y(int NX, int NY, int NZ,
	const float* __restrict__ d_a,
	const float* __restrict__ d_b,
	const float* __restrict__ d_c,
	float* __restrict__ d_d)
{
	int   i, j, k, indg;
	float aa, bb, cc, dd, c[256], d[256];

	//
	// set up indices for main block
	//

	i = threadIdx.x + blockIdx.x*blockDim.x;  // global indices
	k = threadIdx.y + blockIdx.y*blockDim.y;
	indg = i + k*NX*NY;

	if ((i<NX) && (k<NZ)) {

		//
		// forward pass
		//

		bb = 1.0f / d_b[indg];
		cc = bb*d_c[indg];
		dd = bb*d_d[indg];
		c[0] = cc;
		d[0] = dd;

		for (j = 1; j<NY; j++) {
			indg = indg + NX;
			aa = d_a[indg];
			bb = d_b[indg] - aa*cc;
			dd = d_d[indg] - aa*dd;
			bb = 1.0f / bb;
			cc = bb*d_c[indg];
			dd = bb*dd;
			c[j] = cc;
			d[j] = dd;
		}

		//
		// reverse pass
		//

		d_d[indg] = dd;

		for (j = NY - 2; j >= 0; j--) {
			indg = indg - NX;
			dd = d[j] - c[j] * dd;
			d_d[indg] = dd;
		}

	}
}


//
// tri-diagonal solve in z-direction, and update solution
//

__global__ void GPU_adi_z(int NX, int NY, int NZ,
	float* __restrict__ d_u,
	const float* __restrict__ d_a,
	const float* __restrict__ d_b,
	const float* __restrict__ d_c,
	const float* __restrict__ d_d)
{
	int   i, j, k, indg, off;
	float aa, bb, cc, dd, c[256], d[256];

	//
	// set up indices for main block
	//

	i = threadIdx.x + blockIdx.x*blockDim.x;  // global indices
	j = threadIdx.y + blockIdx.y*blockDim.y;
	indg = i + j*NX;
	off = NX*NY;

	if ((i<NX) && (j<NY)) {

		//
		// forward pass
		//

		bb = 1.0f / d_b[indg];
		cc = bb*d_c[indg];
		dd = bb*d_d[indg];
		c[0] = cc;
		d[0] = dd;

		for (k = 1; k<NZ; k++) {
			indg = indg + off;
			aa = d_a[indg];
			bb = d_b[indg] - aa*cc;
			dd = d_d[indg] - aa*dd;
			bb = 1.0f / bb;
			cc = bb*d_c[indg];
			dd = bb*dd;
			c[k] = cc;
			d[k] = dd;
		}

		//
		// reverse pass
		//

		d_u[indg] += dd;

		for (k = NZ - 2; k >= 0; k--) {
			indg = indg - off;
			dd = d[k] - c[k] * dd;
			d_u[indg] += dd;
		}

	}
}




////////////////////////////////////////////////////////////////////////
// declare Gold routine
////////////////////////////////////////////////////////////////////////

void Gold_adi(int, int, int, float, float*, float*, float*, float*,
	float*, float*, float*, float*, float*, float*, float*);

clock_t elapsed_time(clock_t *timer)
{
	return clock() - *timer;
}

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){

	// 'h_' prefix - CPU (host) memory space

	int    NX = 256, NY = 256, NZ = 256, REPEAT = 10, i, j, k, ind, printout = 0;
	float  *h_u1, *h_u2, *h_du,
		*h_ax, *h_bx, *h_cx,
		*h_ay, *h_by, *h_cy,
		*h_az, *h_bz, *h_cz,
		err, lam = 1.0f;

	clock_t timer;
	double elapsed;

	// 'd_' prefix - GPU (device) memory space

	float  *d_u, *d_du,
		*d_ax, *d_bx, *d_cx,
		*d_ay, *d_by, *d_cy,
		*d_az, *d_bz, *d_cz;


	printf("\nGrid dimensions: %d x %d x %d\n", NX, NY, NZ);

	if (NX>256 || NY>256 || NZ>256) {
		printf("No dimension can exceed 256 due to hard-coded array sizes\n");
		return -1;
	}

	// initialise card

	cutilDeviceInit(argc, argv);

	// allocate memory for arrays

	h_u1 = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_u2 = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_du = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_ax = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_bx = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_cx = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_ay = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_by = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_cy = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_az = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_bz = (float *)malloc(sizeof(float)*NX*NY*NZ);
	h_cz = (float *)malloc(sizeof(float)*NX*NY*NZ);

	cudaSafeCall(hipMalloc((void **)&d_u, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_du, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_ax, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_bx, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_cx, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_ay, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_by, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_cy, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_az, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_bz, sizeof(float)*NX*NY*NZ));
	cudaSafeCall(hipMalloc((void **)&d_cz, sizeof(float)*NX*NY*NZ));

	// initialise u1

	for (k = 0; k<NZ; k++) {
		for (j = 0; j<NY; j++) {
			for (i = 0; i<NX; i++) {
				ind = i + j*NX + k*NX*NY;

				if (i == 0 || i == NX - 1 || j == 0 || j == NY - 1 || k == 0 || k == NZ - 1)
					h_u1[ind] = 1.0f;           // Dirichlet b.c.'s
				else
					h_u1[ind] = 0.0f;
			}
		}
	}

	// copy u1 to device

	timer = clock();
	cudaSafeCall(hipMemcpy(d_u, h_u1, sizeof(float)*NX*NY*NZ,
		hipMemcpyHostToDevice));
	cudaSafeCall(hipDeviceSynchronize());
	elapsed = elapsed_time(&timer);
	printf("\nCopy u1 to device: %f (ms) \n", elapsed);

	// Set up the execution configuration

	dim3 dimGrid1(1 + (NX - 1) / BLOCK_X, 1 + (NY - 1) / BLOCK_Y);
	dim3 dimBlock1(BLOCK_X, BLOCK_Y);

	//  dim3 dimGrid2(1+(NX-1)/16, 1+(NY-1)/4);
	//  dim3 dimBlock2(16,4);

	dim3 dimGrid2(1 + (NX - 1) / 32, 1 + (NY - 1) / 4);
	dim3 dimBlock2(32, 4);

	dim3 dimGrid3(NY, NZ);
	dim3 dimBlock3(256);

	// Execute GPU kernel

	double time1 = 0, time2 = 0, time3 = 0, time4 = 0;

	for (i = 1; i <= REPEAT; ++i) {
		GPU_adi_rhs << <dimGrid1, dimBlock1 >> >(NX, NY, NZ,
			lam, d_u, d_du,
			d_ax, d_bx, d_cx,
			d_ay, d_by, d_cy,
			d_az, d_bz, d_cz);
		cudaCheckMsg("GPU_adi_rhs execution failed\n");

		cudaSafeCall(hipDeviceSynchronize());
		time1 += elapsed_time(&timer);

		GPU_adi_x_float4_2 << <dimGrid1, dimBlock1 >> >(NX, NY, NZ,
			(float4*)d_ax, (float4*)d_bx, (float4*)d_cx, (float4*)d_du);

		//  GPU_adi_x<<<dimGrid2, dimBlock2>>>(NX, NY, NZ,
		//                                     d_ax, d_bx, d_cx, d_du);
		//    GPU_adi_x_new<<<dimGrid3, dimBlock3>>>(NX, NY, NZ,
		//                                       d_ax, d_bx, d_cx, d_du);
		cudaCheckMsg("GPU_adi_x execution failed\n");

		cudaSafeCall(hipDeviceSynchronize());
		time2 += elapsed_time(&timer);

		GPU_adi_y << <dimGrid2, dimBlock2 >> >(NX, NY, NZ,
			d_ay, d_by, d_cy, d_du);
		cudaCheckMsg("GPU_adi_y execution failed\n");

		cudaSafeCall(hipDeviceSynchronize());
		time3 += elapsed_time(&timer);

		GPU_adi_z << <dimGrid2, dimBlock2 >> >(NX, NY, NZ, d_u,
			d_az, d_bz, d_cz, d_du);
		cudaCheckMsg("GPUadi_z execution failed\n");

		cudaSafeCall(hipDeviceSynchronize());
		time4 += elapsed_time(&timer);

	}

	printf("\n%dx GPU_adi: %f (ms) %f (ms) %f (ms) %f (ms) \n",
		REPEAT, time1, time2, time3, time4);

	// Read back GPU results

	cudaSafeCall(hipMemcpy(h_u2, d_u, sizeof(float)*NX*NY*NZ,
		hipMemcpyDeviceToHost));
	elapsed = elapsed_time(&timer);
	printf("\nCopy u2 to host: %f (ms) \n", elapsed);

	// print out corner of array

	if (printout) {
		for (k = 0; k<3; k++) {
			for (j = 0; j<8; j++) {
				for (i = 0; i<8; i++) {
					ind = i + j*NX + k*NX*NY;
					printf(" %5.2f ", h_u2[ind]);
				}
				printf("\n");
			}
			printf("\n");
		}
	}

	// Gold treatment

	for (int i = 1; i <= REPEAT; ++i) {
		Gold_adi(NX, NY, NZ,
			lam, h_u1, h_du,
			h_ax, h_bx, h_cx,
			h_ay, h_by, h_cy,
			h_az, h_bz, h_cz);
	}

	elapsed = elapsed_time(&timer);
	printf("\n%dx Gold_adi: %f (ms) \n \n", REPEAT, elapsed);

	// print out corner of array

	if (printout) {
		for (k = 0; k<3; k++) {
			for (j = 0; j<8; j++) {
				for (i = 0; i<8; i++) {
					ind = i + j*NX + k*NX*NY;
					printf(" %5.2f ", h_u1[ind]);
				}
				printf("\n");
			}
			printf("\n");
		}
	}

	// error check

	err = 0.0;

	for (k = 0; k<NZ; k++) {
		for (j = 0; j<NY; j++) {
			for (i = 0; i<NX; i++) {
				ind = i + j*NX + k*NX*NY;
				err += (h_u1[ind] - h_u2[ind])*(h_u1[ind] - h_u2[ind]);
			}
		}
	}

	printf("\nrms error = %f \n", sqrt(err / (float)(NX*NY*NZ)));

	// Release GPU and CPU memory

	cudaSafeCall(hipFree(d_u));
	cudaSafeCall(hipFree(d_du));
	cudaSafeCall(hipFree(d_ax));
	cudaSafeCall(hipFree(d_bx));
	cudaSafeCall(hipFree(d_cx));
	cudaSafeCall(hipFree(d_ay));
	cudaSafeCall(hipFree(d_by));
	cudaSafeCall(hipFree(d_cy));
	cudaSafeCall(hipFree(d_az));
	cudaSafeCall(hipFree(d_bz));
	cudaSafeCall(hipFree(d_cz));
	free(h_u1);
	free(h_u2);
	free(h_du);
	free(h_ax);
	free(h_bx);
	free(h_cx);
	free(h_ay);
	free(h_by);
	free(h_cy);
	free(h_az);
	free(h_bz);
	free(h_cz);

	hipDeviceReset();

	system("pause");
}
