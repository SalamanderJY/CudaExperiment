#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include "cutil_inline.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;


////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////


__global__ void pathcalc(float *d_z, float *d_v)
{
	float s1, s2, y1, y2, payoff;

	// move array pointers to correct position

	// version 1
	//d_z = d_z + threadIdx.x + 2 * N*blockIdx.x*blockDim.x;

	// version 2
	d_z = d_z + 2 * N * threadIdx.x + 2 * N * blockIdx.x * blockDim.x;

	d_v = d_v + threadIdx.x + blockIdx.x*blockDim.x;

	// path calculation

	s1 = 1.0f;
	s2 = 1.0f;

	for (int n = 0; n < N; n++) {
		y1 = (*d_z);
		// version 1
		//d_z += blockDim.x;      // shift pointer to next element
		// version 2
		d_z += 1; 

		y2 = rho*y1 + alpha*(*d_z);
		// version 1
		//d_z += blockDim.x;      // shift pointer to next element
		// version 2
		d_z += 1; 

		s1 = s1*(con1 + con2*y1);
		s2 = s2*(con1 + con2*y2);
	}

	// put payoff value into device array

	payoff = 0.0f;
	if (fabs(s1 - 1.0f) < 0.1f && fabs(s2 - 1.0f) < 0.1f) payoff = exp(-r*T);

	*d_v = payoff;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main_monte(int argc, char* argv[]) {

	int     NPATH = 960000, h_N = 100;
	float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
	float  *h_v, *d_v, *d_z;
	double  sum1, sum2;

	//double timer, elapsed;  
	clock_t timer;   // for counting the CPU time
	double elapsed;	 // elapsed time	

	hiprandGenerator_t gen;

	// initialise card

	cutilDeviceInit(argc, argv);

	// allocate memory on host and device

	h_v = (float *)malloc(sizeof(float)*NPATH);

	cudaSafeCall(hipMalloc((void **)&d_v, sizeof(float)*NPATH));
	cudaSafeCall(hipMalloc((void **)&d_z, sizeof(float)* 2 * h_N*NPATH));

	// define constants and transfer to GPU

	h_T = 1.0f;
	h_r = 0.05f;
	h_sigma = 0.1f;
	h_rho = 0.5f;
	h_alpha = sqrt(1.0f - h_rho*h_rho);
	h_dt = 1.0f / h_N;
	h_con1 = 1.0f + h_r*h_dt;
	h_con2 = sqrt(h_dt)*h_sigma;

	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(h_N)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(T), &h_T, sizeof(h_T)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(r), &h_r, sizeof(h_r)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(sigma), &h_sigma, sizeof(h_sigma)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(rho), &h_rho, sizeof(h_rho)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(alpha), &h_alpha, sizeof(h_alpha)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dt), &h_dt, sizeof(h_dt)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)));

	// random number generation

	timer = clock();  // initialise timer

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
	hiprandGenerateNormal(gen, d_z, 2 * h_N*NPATH, 0.0f, 1.0f);

	cudaSafeCall(hipDeviceSynchronize());

	elapsed = elapsed_time(&timer);
	printf("\nCURAND normal RNG execution time (ms): %f ,   samples/sec: %e \n",
		elapsed, 2.0*h_N*NPATH / elapsed);

	// execute kernel and time it

	pathcalc << <NPATH / 64, 64 >> >(d_z, d_v);
	cudaCheckMsg("pathcalc execution failed\n");
	cudaSafeCall(hipDeviceSynchronize());

	elapsed = elapsed_time(&timer);
	printf("Monte Carlo kernel execution time (ms): %f \n", elapsed);

	// copy back results

	cudaSafeCall(hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
		hipMemcpyDeviceToHost));

	// compute average

	sum1 = 0.0;
	sum2 = 0.0;
	for (int i = 0; i < NPATH; i++) {
		sum1 += h_v[i];
		sum2 += h_v[i] * h_v[i];
	}

	printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
		sum1 / NPATH, sqrt((sum2 / NPATH - (sum1 / NPATH)*(sum1 / NPATH)) / NPATH));

	// Tidy up library

	hiprandDestroyGenerator(gen);

	// Release memory and exit cleanly

	free(h_v);
	cudaSafeCall(hipFree(d_v));
	cudaSafeCall(hipFree(d_z));

	// CUDA exit -- needed to flush printf write buffer

	hipDeviceReset();
	system("pause");

	return 0;
}