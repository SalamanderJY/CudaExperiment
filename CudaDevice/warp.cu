#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void GetThreadId(
	unsigned int *block, 
	unsigned int *thread, 
	unsigned int *warp, 
	unsigned int *calc_thread, 
	unsigned int *clocks)
{
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	block[thread_id] = blockIdx.x;
	thread[thread_id] = threadIdx.x;
	warp[thread_id] = threadIdx.x / warpSize;
	calc_thread[thread_id] = thread_id;
	clocks[thread_id] = clock();
}
#define ArraySize Arraysize * sizeof(unsigned int)

int warp_main(int argc, char* argv[])
{
	unsigned int block_nums, thread_nums;
	printf("Input blocknums and threadnums\n");
	scanf("%d%d", &block_nums, &thread_nums);
	const unsigned int Arraysize = block_nums * thread_nums;

	unsigned int *gpu_block, *gpu_thread, *gpu_warp, *gpu_calc, *gpu_clock;

	hipMalloc((void**)&gpu_block, ArraySize);
	hipMalloc((void**)&gpu_thread, ArraySize);
	hipMalloc((void**)&gpu_warp, ArraySize);
	hipMalloc((void**)&gpu_calc, ArraySize);
	hipMalloc((void**)&gpu_clock, ArraySize);

	GetThreadId << <block_nums, thread_nums >> >(gpu_block, gpu_thread, gpu_warp, gpu_calc, gpu_clock);

	unsigned int* cpu_block = new unsigned int[Arraysize];
	unsigned int* cpu_thread = new unsigned int[Arraysize];
	unsigned int* cpu_warp = new unsigned int[Arraysize];
	unsigned int* cpu_calc = new unsigned int[Arraysize];
	unsigned int* cpu_clock = new unsigned int[Arraysize];

	hipMemcpy(cpu_block, gpu_block, ArraySize, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_thread, gpu_thread, ArraySize, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_warp, gpu_warp, ArraySize, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_calc, gpu_calc, ArraySize, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_clock, gpu_clock, ArraySize, hipMemcpyDeviceToHost);

	hipFree(gpu_block); 
	hipFree(gpu_thread);
	hipFree(gpu_warp); 
	hipFree(gpu_calc); 
	hipFree(gpu_clock);

	for (int i = 0; i<Arraysize; i++)
	{
		// output the paramters, ps:cpu_clock[i] - cpu_clock[0] is represented of the difference between the run time of each thread and start thread,it is easy to calculate and observe.
		printf("Calculated Thread: %3u- Block: %3u- Warp: %3u- Thread: %3u- Time: %3u\n", cpu_calc[i], cpu_block[i], cpu_warp[i], cpu_thread[i], cpu_clock[i] - cpu_clock[0]);
	}

	delete cpu_block;
	delete cpu_thread;
	delete cpu_warp;
	delete cpu_calc;
	delete cpu_clock;

	system("pause");
	return 0;
}

