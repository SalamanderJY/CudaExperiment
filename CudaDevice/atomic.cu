#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <device_atomic_functions.hpp>

#include <stdlib.h>

#define TPB 64
#define ATOMIC 1

#define N 1024

__global__ void dotKernel(int *d_res, const int *d_a, const int *d_b, int n)
{
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= n) return;
	const int s_idx = threadIdx.x;

	__shared__ int s_prod[TPB];
	s_prod[s_idx] = d_a[idx] * d_b[idx];
	__syncthreads();

	if (s_idx == 0)
	{
		int blockSum = 0;
		for (int j = 0; j < blockDim.x; ++j)
		{
			blockSum += s_prod[j];
		}

		printf("Block_%d, blockSum = %d\n", blockIdx.x, blockSum);

		if (ATOMIC)
		{
			atomicAdd(d_res, blockSum);
		}
		else
		{
			*d_res += blockSum;
		}
	}
}

void dotLauncher(int *res, const int *a, const int *b, int n)
{
	int *d_res;
	int *d_a = 0;
	int *d_b = 0;

	hipMalloc(&d_res, sizeof(int));
	hipMalloc(&d_a, n*sizeof(int));
	hipMalloc(&d_b, n*sizeof(int));

	hipMemset(d_res, 0, sizeof(int));
	hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);

	dotKernel << <(n + TPB - 1) / TPB, TPB >> >(d_res, d_a, d_b, n);
	hipMemcpy(res, d_res, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_res);
	hipFree(d_a);
	hipFree(d_b);
}

int atomic_main(int argc, char* argv[])
{
	int cpu_res = 0;
	int gpu_res = 0;
	int *a = (int*)malloc(N*sizeof(int));
	int *b = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
	{
		a[i] = 1;
		b[i] = 1;
	}

	for (int i = 0; i < N; ++i)
	{
		cpu_res += a[i] * b[i];
	}
	printf("cpu result = %d\n", cpu_res);

	dotLauncher(&gpu_res, a, b, N);
	printf("gpu_result = %d\n", gpu_res);

	free(a);
	free(b);

	system("pause");

	return 0;
}